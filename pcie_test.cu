#include <stdio.h>
#include <hip/hip_runtime.h>
#define times 256
// CUDA 初始化
void cudaInitialize(int deviceId) {
    hipError_t err = hipSetDevice(deviceId);
    if (err != hipSuccess) {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
        exit(EXIT_FAILURE);
    }
}

// 计算带宽的函数
void calculateBandwidth(size_t bytes, float timeMs) {
    float bandwidth =times * (bytes / 1e6) / (timeMs / 1e3); // MB/s
    printf("Bandwidth: %f MB/s\n", bandwidth);
}

int main() {
    // CUDA 初始化
    cudaInitialize(0);

    // 数据大小
    size_t dataSize = 1536 * 1024 * 1024;
    // 分配主机内存
    char *h_data = (char *)malloc(dataSize);

    // 分配设备内存
    char *d_data;
    hipMallocPitch((void **)&d_data, dataSize);

    // 填充主机内存
    #pragma omp parallel for
    for (size_t i = 0; i < dataSize; ++i) {
        h_data[i] = (char)i;
    }

    // 记录开始时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // 将数据从主机复制到设备
    for(int i=0;i<times;i++)
    {
    hipMemcpy(d_data, h_data, dataSize, hipMemcpyHostToDevice);
    }
    // 记录结束时间
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // 计算时间
    float elapsedTimeMs = 0.0f;
    hipEventElapsedTime(&elapsedTimeMs, start, stop);

    // 计算带宽
    calculateBandwidth(dataSize, elapsedTimeMs);

    // 释放资源
    hipFree(d_data);
    free(h_data);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
